#include "hip/hip_runtime.h"
#include <stdio.h>
#include "bfs.h"
#include "util.h"
#include "stats.h"
#include "cuda_util.h"
#include "graph_util.h"

#define ENABLE_CPU_RECURSIVE

static stats_t stats;

void validateArrays(node_t n, unsigned int *array1, unsigned int *array2, const char *message){
	for (node_t node=0; node<n;node++){
		if (array1[node]!=array2[node]){
			printf("ERROR: validation error at %llu: %s !\n", node, message);
			break;
		}
	}
}


unsigned get_num_levels(graph_t graph){
	unsigned level = 0;
	for (node_t n=0; n < graph.num_nodes; n++){
		if (graph.levelArray[n]!=UNDEFINED) level = max(level,graph.levelArray[n]);
	}
	return (level+1);
}

int main(int argc, char *argv[])
{
	//init_device(DEVICE);
	double time;
	graph_t graph;
	char *filename = "test/graph.dot"; //unused at the moment...	
	unsigned dataset_num = 0;

	//process the input parameters
	if ( argc==2 ) {
		dataset_num = atoi(argv[1]);
	}
	else {
		printf("Usage: run_bfs [dataset] < [path to graph data file]\n");
		printf("dataset: 0 - DIMACS9\n");
		printf("         1 - DIMACS10\n");
		printf("         2 - SLNDC\n");
		exit(0);
	}

	//read graph datafile and convert it to CSR
	switch(dataset_num){
		case 0: readInputDIMACS9(&graph); break;
		case 1: readInputDIMACS10(&graph); break;
		case 2: readInputSLNDC(&graph); break;
		default: printf("Wrong code for dataset\n"); break;
	}

	//starts execution
	printf("\n===MAIN=== :: [num_nodes,num_edges] = %u, %u\n", graph.num_nodes, graph.num_edges);

	stats.num_nodes = graph.num_nodes;
	stats.num_edges = graph.num_edges;

	//compute bfs on CPU iteratively
	time = gettime_ms();
	if (graph.num_nodes!=0) bfs(&graph);
	stats.cpu_time_it=gettime_ms()-time;
	printf("===> CPU #1 time to compute bfs = %.2f ms.\n",gettime_ms()-time);

	stats.levels = get_num_levels(graph);

#ifdef ENABLE_CPU_RECURSIVE
	//compute bfs on CPU recursively
	time = gettime_ms();
	if (graph.num_nodes!=0) bfs_rec(&graph);
	stats.cpu_time_rec=gettime_ms()-time;
	printf("===> CPU #2 time to compute bfs recursively = %.2f ms.\n",gettime_ms()-time);

	validateArrays(graph.num_nodes, graph.levelArray, graph.levelArray_rec, "CPU bfs rec");
#endif

	//compute bfs on GPU 
	if (graph.num_nodes!=0) bfs_gpu(&graph, &stats);

	validateArrays(graph.num_nodes, graph.levelArray, graph.levelArray_gpu, "GPU #1 bfs");
	validateArrays(graph.num_nodes, graph.levelArray, graph.levelArray_gpu_np, "GPU #2 bfs np");
	validateArrays(graph.num_nodes, graph.levelArray, graph.levelArray_gpu_np_hier, "GPU #3 bfs np hier");

	//write stats file
	FILE *file = fopen("stats.txt", "w+");
	print_stats(file, stats);
	fclose(file);
	
	delete_graph(&graph);

	return(0);
}
